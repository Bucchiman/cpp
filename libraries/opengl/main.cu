#include "hip/hip_runtime.h"
/*
 * FileName:     main
 * Author:       8ucchiman
 * CreatedDate:  2023-05-17 16:45:17
 * LastModified: 2023-02-26 13:30:39 +0900
 * Reference:    8ucchiman.jp
 * Description:  ---
 */


#include <stdio.h>
#include <GL/glut.h>
//#include "glew.h"

#define MACRO
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s: %d)\n", \
                    msg, hipGetErrorString(__err), \
                    __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void hoge(void *arguments) {

}

void display(void) {
}


#ifdef MACRO
int main(int argc, char* argv[]){
    glutInit();
    //glewCreate
    return 0;
}
#endif

